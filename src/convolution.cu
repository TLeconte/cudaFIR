#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

extern "C" {
#include "cudaFIR.h"
}

static __global__ void cufftComplexPointwiseMul(hipfftComplex *, const hipfftComplex *, const hipfftComplex *);
static __global__ void AddOverlapScale(hipfftReal *a, const hipfftReal *b, float scale);
static __global__ void ShiftAndPad(const hipfftReal *a, hipfftReal *b, int shift ,int len);

static const char *filter_FSstr[NBFILTER]={ "-44k", "-48k", "-88k", "-96k" , "-176k", "-192k", "-352k" , "-384k", "-705k" , "-768k" };

static  hipfftReal *d_signal;
static  hipfftComplex *d_signal_fft;
static  hipfftComplex *d_filter_fft[NBFILTER];
static  hipfftComplex *d_tmp_fft;
static  hipfftReal *d_tmp_signal;
static  hipfftReal *d_convolved_signal[2];
static  hipfftHandle fplan,bplan;

static int bk=0,nbk;

#define NBTHREADS 256
#define NBCHANN (cvparam->nbch)
#define PART_SIZE (cvparam->partsz)
#define FFT_SIZE (2*PART_SIZE)
#define FFT_CSIZE (((FFT_SIZE/2+512)/512)*512)

static int addFilter(conv_param_t *cvparam, float *h_filter, int nf) { 

  hipMalloc((void **)(&(d_filter_fft[nf])), sizeof(hipfftComplex)*FFT_CSIZE*NBCHANN*cvparam->nbpart[nf]);

  // compute fft filter parts
  for(int n=0; n < cvparam->nbpart[nf] ; n++ ) {
  	// Copy host memory to device
  	hipMemcpy(d_signal, &(h_filter[n*NBCHANN*PART_SIZE]), sizeof(float)*PART_SIZE*NBCHANN, hipMemcpyHostToDevice);
  	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "cudaFIR: Memcpy filter\n");
		return -1;
  	}

  	hipfftExecR2C(fplan, d_signal, &(d_filter_fft[nf][n*NBCHANN*FFT_CSIZE]));
  	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "cudaFIR: ExecR2C %d\n",n);
		return -1;
  	}
  }

  return 0;
}

static int readFilter(char *filterpath,conv_param_t *cvparam,int nf)
{
        FILE *fd;
        int size;
        float *filter;

        fd=fopen(filterpath,"r");
        if(fd==NULL) return -1;


	fseek(fd, 0, SEEK_END);
	size=ftell(fd)/sizeof(float);
	rewind(fd);
	cvparam->nbpart[nf] = (size/cvparam->nbch+cvparam->partsz-1)/cvparam->partsz;
       	fprintf(stderr,"filter %s sz:%d npart:%d nf:%d\n",filterpath,size,cvparam->nbpart[nf],nf);

        size=cvparam->nbch*cvparam->partsz*cvparam->nbpart[nf];

        filter=(float*)calloc(size,sizeof(float));
        if(filter==NULL) {
                fclose(fd);
                return -1;
        }

        if(fread(filter,sizeof(float),size,fd)!=size) {
       		fprintf(stderr,"cudaFIR read filter error \n");
                free(filter);
                fclose(fd);
                return -1;
        }
        fclose(fd);

        if(addFilter(cvparam,filter,nf)) {
                free(filter);
                return -1;
        }

        free(filter);
        return 0;
}


int initConvolve(conv_param_t *cvparam,char *filterpathprefix) { 
  int n,max;
  char *filterpath;

  if(cvparam->partsz%NBTHREADS) {
	fprintf(stderr, "cudaFIR : partsz must be a multiple of %d\n",NBTHREADS);
	return -1;
  }

  hipHostAlloc(&(cvparam->inoutbuff),sizeof(float)*FFT_SIZE*NBCHANN,0);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR  : cudaAlllocHost error\n");
	return -1;
  }

  // Allocate device memory 
  hipMalloc((void **)(&d_signal), sizeof(hipfftReal)*FFT_SIZE*NBCHANN);
  hipMalloc((void **)(&d_signal_fft), sizeof(hipfftComplex)*FFT_CSIZE*NBCHANN);
  hipMalloc((void **)(&d_tmp_fft), sizeof(hipfftComplex)*FFT_CSIZE*NBCHANN);
  hipMalloc((void **)(&d_tmp_signal), sizeof(hipfftReal)*FFT_SIZE*NBCHANN);

  // CUFFT plan 
  int inembed=1;
  int onembed=1;
  int fftsz=FFT_SIZE; 

  hipfftPlanMany(&fplan,1, &fftsz, &inembed, NBCHANN, inembed, &onembed, NBCHANN, onembed, HIPFFT_R2C, NBCHANN);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: Plan1d\n");
	return -1;
  }
  hipfftPlanMany(&bplan,1, &fftsz, &inembed, NBCHANN, inembed, &onembed, NBCHANN, onembed, HIPFFT_C2R, NBCHANN);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: Plan1d\n");
	return -1;
  }

  // pad signals
  hipMemset(d_signal+PART_SIZE*NBCHANN, 0, sizeof(hipfftReal) * PART_SIZE * NBCHANN);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: MulAndScale\n");
	return -1;
  }

  max=0;
  filterpath=(char*)malloc(strlen(filterpathprefix)+16);
  for(n=0;n<NBFILTER;n++) {
      d_filter_fft[n]=NULL;
      cvparam->nbpart[n]=0;

      strcpy(filterpath,filterpathprefix);
      strcat(filterpath,filter_FSstr[n]);
      strcat(filterpath,".raw");
      readFilter(filterpath,cvparam,n);
      if(cvparam->nbpart[n]>max) max=cvparam->nbpart[n];
  }
  free(filterpath);

  fprintf(stderr,"filper max :%d\n",max);

  hipMalloc((void **)(&(d_convolved_signal[0])), sizeof(hipfftReal)*NBCHANN*(max+1)*PART_SIZE);
  hipMalloc((void **)(&(d_convolved_signal[1])), sizeof(hipfftReal)*NBCHANN*(max+1)*PART_SIZE);
  hipMemset(d_convolved_signal[0], 0, sizeof(hipfftReal)*NBCHANN*(max+1)*PART_SIZE);
  hipMemset(d_convolved_signal[1], 0, sizeof(hipfftReal)*NBCHANN*(max+1)*PART_SIZE);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: Memset\n");
	return -1;
  }

  return 0;
}


void waitConvolve(void)
{
	hipStreamSynchronize(0);
}

#define FILTER_NPART (cvparam->nbpart[cvparam->nf])
int cudaConvolve(conv_param_t *cvparam) { 

  hipMemcpyAsync(d_signal,cvparam->inoutbuff, sizeof(float)*PART_SIZE*NBCHANN, hipMemcpyHostToDevice,0);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: fail to Memcpy d_signal\n");
	return -1;
  }

  /* signal FFT */
  hipfftExecR2C(fplan, d_signal, d_signal_fft);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: ExecR2C\n");
	return -1;
  }

  for(int n=0; n < FILTER_NPART ; n++ ) {
	/*  signal fft * filter part fft */
  	cufftComplexPointwiseMul<<<(FFT_CSIZE*NBCHANN/NBTHREADS),NBTHREADS>>>(d_tmp_fft, d_signal_fft, &(d_filter_fft[cvparam->nf][n*FFT_CSIZE*NBCHANN]));
  	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "cudaFIR: Mul\n");
		return -1;
  	}

	/* ifft */
  	hipfftExecC2R(bplan, d_tmp_fft, d_tmp_signal);
  	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "cudaFIR: ExecC2R\n");
		return -1;
  	}

	/* overlap and add result */
  	AddOverlapScale<<<FFT_SIZE*NBCHANN/NBTHREADS,NBTHREADS>>>(&(d_convolved_signal[bk][n*PART_SIZE*NBCHANN]), d_tmp_signal, 1.0/(float)FFT_SIZE);
  	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "cudaFIR: AddOverlap\n");
		return -1;
  	}

  }

  /* result device to host copy  */
  hipMemcpyAsync(cvparam->inoutbuff, d_convolved_signal[bk], sizeof(float) * PART_SIZE * NBCHANN, hipMemcpyDeviceToHost,0);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: fail to Memcpy d_convolved\n");
	return -1;
  }

  // shift
  nbk=bk^1;
  ShiftAndPad<<<(((FILTER_NPART+1)*PART_SIZE)*NBCHANN/NBTHREADS),NBTHREADS>>>(d_convolved_signal[bk], d_convolved_signal[nbk],PART_SIZE*NBCHANN, FILTER_NPART*PART_SIZE*NBCHANN);
  if (hipGetLastError() != hipSuccess){
	fprintf(stderr, "cudaFIR: Shift kernel error\n");
	return -1;
    }
  bk=nbk;

  return 0;
}

void freeFilter(void)
{
  int n;

  // Destroy CUFFT context
  hipfftDestroy(fplan);
  hipfftDestroy(bplan);

  hipFree(d_signal);
  hipFree(d_signal_fft);
  for(n=0;n<NBFILTER;n++)
  	if(d_filter_fft[n])
  		hipFree(d_filter_fft[n]);
  hipFree(d_tmp_fft);
  hipFree(d_tmp_signal);
  hipFree(d_convolved_signal[0]);
  hipFree(d_convolved_signal[1]);
}

////////////////////////////////////////////////////////////////////////////////
// kernels
////////////////////////////////////////////////////////////////////////////////
static __global__ void cufftComplexPointwiseMul(hipfftComplex *a, const hipfftComplex *b, const hipfftComplex *c) {
  const int tID = blockIdx.x * blockDim.x + threadIdx.x;

    a[tID] = hipCmulf(b[tID], c[tID]);
}

static __global__ void AddOverlapScale(hipfftReal *a, const hipfftReal *b, float scale) {
  const int tID = blockIdx.x * blockDim.x + threadIdx.x;

    a[tID] = a[tID]+ b[tID]*scale;
}

static __global__ void ShiftAndPad(const hipfftReal *a, hipfftReal *b, int shift ,int len) {
  const int tID = blockIdx.x * blockDim.x + threadIdx.x;

  if(tID<len)
    b[tID] = a[tID+shift];
  else
    b[tID]=0;

}
